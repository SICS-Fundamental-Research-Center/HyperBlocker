/**
 * @section DESCRIPTION
 *
 * Run HyperBlocker
 */

#include <hip/hip_runtime.h>
#include <>
#include <gflags/gflags.h>

#include <fstream>
#include <iostream>
#include <list>
#include <utility>

#include "core/common/types.h"
#include "core/common/yaml_config.h"
#include "core/gpu/global_func.cuh"
#include "core/hyperblocker.cuh"

DEFINE_string(data_l, "", "input dir 2.");
DEFINE_string(data_r, "", "input dir 1.");

DEFINE_string(rule_dir, "", "root path of rules.");
DEFINE_string(o, "", "output path.");
DEFINE_string(sep, ",", "separator to split a line of csv file.");
DEFINE_bool(read_header, false, "whether to read header of csv.");
DEFINE_uint64(n_partitions, 1, "number of partitions.");
DEFINE_uint64(prefix_hash_predicate_index, INT_MAX, "number of partitions.");

int main(int argc, char *argv[]) {
  gflags::ParseCommandLineFlags(&argc, &argv, true);

  sics::hyperblocker::core::HyperBlocker hb(
      FLAGS_rule_dir, FLAGS_data_l, FLAGS_data_r, FLAGS_o, FLAGS_n_partitions,
      FLAGS_prefix_hash_predicate_index, FLAGS_sep);
  // hb.ShowDeviceProperties();
  // hb.Initialize();
  hb.Run();

  gflags::ShutDownCommandLineFlags();
  return EXIT_SUCCESS;
}
